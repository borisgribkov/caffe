#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/add_angular_margin_layer.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void AddAngularMarginForward(const int n, const int dim, const Dtype* label,
                                                 const Dtype* bottom_data, Dtype* top_data, Dtype angle) {
    Dtype cos_m = cosf(angle);
    Dtype sin_m = sinf(angle);
    Dtype threshold = cosf(M_PI - angle);
    Dtype mm = sinf(M_PI - angle) * angle;
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      Dtype bottom_val = bottom_data[index * dim + gt];
      if(bottom_val > threshold) {
        Dtype sin_t = sqrtf(1 - bottom_val * bottom_val);
        top_data[index * dim + gt] = bottom_val * cos_m - sin_t * sin_m;
      } else {
        top_data[index * dim + gt] = bottom_val - mm;
      }
    }
  }

  template <typename Dtype>
  __global__ void AddAngularMarginBackward(const int n, const int dim, const Dtype* label,
                                                 const Dtype* bottom_data, Dtype* bottom_diff, Dtype angle) {
    Dtype cos_m = cosf(angle);
    Dtype sin_m = sinf(angle);
    Dtype threshold = cosf(M_PI - angle);
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      Dtype bottom_val = bottom_data[index * dim + gt];
      if(bottom_val > threshold) {
        Dtype sin_t = sqrtf(1 - bottom_val * bottom_val);
        bottom_diff[index * dim + gt] *= cos_m + sin_m * bottom_val / sin_t;
      }
    }
  }

  template <typename Dtype>
  void AddAngularMarginLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;
    caffe_copy(count, bottom_data, top_data);
    // NOLINT_NEXT_LINE(whitespace/operators)
    AddAngularMarginForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, label_data, bottom_data, top_data, angle_);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  void AddAngularMarginLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                     const vector<bool>& propagate_down,
                                                     const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) { return; }
    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, top_diff, bottom_diff);
    // NOLINT_NEXT_LINE(whitespace/operators)
    AddAngularMarginBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, label_data, bottom_data, bottom_diff, angle_);
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(AddAngularMarginLayer);
}  // namespace caffe